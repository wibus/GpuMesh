#include "hip/hip_runtime.h"
#include "Base.cuh"
#include <Smoothers/AbstractSmoother.h>


// Vertex Accum
__device__ bool assignAverage(uint vId, vec3& pos);
__device__ void reinitAccum(uint vId);

// Smoothing Helper
__device__ uint getInvocationVertexId();
__device__ bool isSmoothableVertex(uint vId);
__device__ float patchQuality(uint vId);


__global__ void updateVerticesCudaMain()
{
    uint vId = getInvocationVertexId();

    if(isSmoothableVertex(vId))
    {
        vec3 pos = verts[vId].p;
        vec3 posPrim = pos;

        if(assignAverage(vId, posPrim))
        {
            Topo topo = topos[vId];
            if(topo.type > 0)
                posPrim = snapToBoundary(topo.type, posPrim);

            float prePatchQuality =
                patchQuality(vId);

            verts[vId].p = posPrim;

            float patchQualityPrime =
                patchQuality(vId);

            if(patchQualityPrime < prePatchQuality)
                verts[vId].p = pos;
        }

        reinitAccum(vId);
    }
}


// CUDA Drivers
void setupCudaIndependentDispatch(const IndependentDispatch& dispatch);
void updateCudaSmoothedElementsVertices(
        const IndependentDispatch& dispatch,
        size_t workgroupSize)
{
    setupCudaIndependentDispatch(dispatch);

    cudaCheckErrors("CUDA error before vertices update");
    updateVerticesCudaMain<<<dispatch.workgroupCount, workgroupSize>>>();
    cudaCheckErrors("CUDA error in vertices update");

    hipDeviceSynchronize();
}
