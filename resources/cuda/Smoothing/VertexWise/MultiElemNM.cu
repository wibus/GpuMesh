#include "hip/hip_runtime.h"
#include "Base.cuh"

#include <DataStructures/NodeGroups.h>


#define NODE_THREAD_COUNT uint(4)
#define ELEMENT_THREAD_COUNT uint(8)

#define ELEMENT_PER_THREAD_COUNT uint(96 / ELEMENT_THREAD_COUNT)

#define MIN_MAX 2147483647


namespace menm
{
    __constant__ float VALUE_CONVERGENCE;
    __constant__ int SECURITY_CYCLE_COUNT;
    __constant__ float LOCALE_SIZE_TO_NODE_SHIFT;
    __constant__ float ALPHA;
    __constant__ float BETA;
    __constant__ float GAMMA;
    __constant__ float DELTA;

    __shared__ int patchMin[NODE_THREAD_COUNT];
    __shared__ float patchMean[NODE_THREAD_COUNT];
}

using namespace menm;


// Smoothing Helper
__device__ uint getInvocationVertexId();
__device__ bool isSmoothableVertex(uint vId);
__device__ float computeLocalElementSize(uint vId);

__device__ float multiElemPatchQuality(
        uint nBeg, uint nEnd,
        uint neigElemCount,
        const vec3& pos)
{
    uint nId = threadIdx.x;

    patchMin[nId] = MIN_MAX;
    patchMean[nId] = 0.0;

    __syncthreads();


    for(uint e = nBeg; e < nEnd; ++e)
    {
        NeigElem& elem = neigElems[e];
        vec3 vertPos[HEX_VERTEX_COUNT];

        float qual = 0.0;
        switch(elem.type)
        {
        case TET_ELEMENT_TYPE :
            vertPos[0] = verts[tets[elem.id].v[0]].p;
            vertPos[1] = verts[tets[elem.id].v[1]].p;
            vertPos[2] = verts[tets[elem.id].v[2]].p;
            vertPos[3] = verts[tets[elem.id].v[3]].p;
            vertPos[elem.vId] = pos;
            qual = (*tetQualityImpl)(vertPos, tets[elem.id]);
            break;

        case PRI_ELEMENT_TYPE :
            vertPos[0] = verts[pris[elem.id].v[0]].p;
            vertPos[1] = verts[pris[elem.id].v[1]].p;
            vertPos[2] = verts[pris[elem.id].v[2]].p;
            vertPos[3] = verts[pris[elem.id].v[3]].p;
            vertPos[4] = verts[pris[elem.id].v[4]].p;
            vertPos[5] = verts[pris[elem.id].v[5]].p;
            vertPos[elem.vId] = pos;
            qual = (*priQualityImpl)(vertPos, pris[elem.id]);
            break;

        case HEX_ELEMENT_TYPE :
            vertPos[0] = verts[hexs[elem.id].v[0]].p;
            vertPos[1] = verts[hexs[elem.id].v[1]].p;
            vertPos[2] = verts[hexs[elem.id].v[2]].p;
            vertPos[3] = verts[hexs[elem.id].v[3]].p;
            vertPos[4] = verts[hexs[elem.id].v[4]].p;
            vertPos[5] = verts[hexs[elem.id].v[5]].p;
            vertPos[6] = verts[hexs[elem.id].v[6]].p;
            vertPos[7] = verts[hexs[elem.id].v[7]].p;
            vertPos[elem.vId] = pos;
            qual = (*hexQualityImpl)(vertPos, hexs[elem.id]);
            break;
        }

        atomicMin(&patchMin[nId], qual * MIN_MAX);
        atomicAdd(&patchMean[nId], 1.0 / qual);
    }

    __syncthreads();


    float patchQual = 0.0;

    if(patchMin[nId] <= 0.0)
        patchQual = patchMin[nId] / float(MIN_MAX);
    else
        patchQual = neigElemCount / patchMean[nId];

    return patchQual;
}


// ENTRY POINT //
__device__ void multiElemNMSmoothVert(uint vId)
{
    uint eId = threadIdx.y;

    Topo topo = topos[vId];
    uint neigElemCount = topo.neigElemCount;
    uint nBeg = topo.neigElemBase + (eId * neigElemCount) / ELEMENT_THREAD_COUNT;
    uint nEnd = topo.neigElemBase + ((eId+1) * neigElemCount) / ELEMENT_THREAD_COUNT;


    // Compute local element size
    float localSize = computeLocalElementSize(vId);

    // Initialize node shift distance
    float nodeShift = localSize * LOCALE_SIZE_TO_NODE_SHIFT;

    vec3 pos = verts[vId].p;
    vec4 vo(pos, multiElemPatchQuality(nBeg, nEnd, neigElemCount, pos));

    vec4 simplex[TET_VERTEX_COUNT] = {
        vec4(pos + vec3(nodeShift, 0, 0), 0),
        vec4(pos + vec3(0, nodeShift, 0), 0),
        vec4(pos + vec3(0, 0, nodeShift), 0),
        vo
    };

    int cycle = 0;
    bool reset = false;
    bool terminated = false;
    while(!terminated)
    {
        for(uint p=0; p < TET_VERTEX_COUNT-1; ++p)
        {
            // Since 'pos' is a reference on vertex's position
            // modifing its value here should be seen by the evaluator
            verts[vId].p = vec3(simplex[p]);

            // Compute patch quality
            simplex[p] = vec4(verts[vId].p, multiElemPatchQuality(nBeg, nEnd, neigElemCount, verts[vId].p));
        }

        // Mini bubble sort
        if(simplex[0].w > simplex[1].w)
            swap(simplex[0], simplex[1]);
        if(simplex[1].w > simplex[2].w)
            swap(simplex[1], simplex[2]);
        if(simplex[2].w > simplex[3].w)
            swap(simplex[2], simplex[3]);
        if(simplex[0].w > simplex[1].w)
            swap(simplex[0], simplex[1]);
        if(simplex[1].w > simplex[2].w)
            swap(simplex[1], simplex[2]);
        if(simplex[0].w > simplex[1].w)
            swap(simplex[0], simplex[1]);


        for(; cycle < SECURITY_CYCLE_COUNT; ++cycle)
        {
            // Centroid
            vec3 c = 1/3.0f * (
                vec3(simplex[1]) +
                vec3(simplex[2]) +
                vec3(simplex[3]));

            float f = 0.0;

            // Reflect
            verts[vId].p = c + ALPHA*(c - vec3(simplex[0]));
            float fr = f = multiElemPatchQuality(nBeg, nEnd, neigElemCount, verts[vId].p);

            vec3 xr = verts[vId].p;

            // Expand
            if(simplex[3].w < fr)
            {
                verts[vId].p = c + GAMMA*(verts[vId].p - c);
                float fe = f = multiElemPatchQuality(nBeg, nEnd, neigElemCount, verts[vId].p);

                if(fe <= fr)
                {
                    verts[vId].p = xr;
                    f = fr;
                }
            }
            // Contract
            else if(simplex[1].w >= fr)
            {
                // Outside
                if(fr > simplex[0].w)
                {
                    verts[vId].p = c + BETA*(vec3(xr) - c);
                    f = multiElemPatchQuality(nBeg, nEnd, neigElemCount, verts[vId].p);
                }
                // Inside
                else
                {
                    verts[vId].p = c + BETA*(vec3(simplex[0]) - c), 0;
                    f = multiElemPatchQuality(nBeg, nEnd, neigElemCount, verts[vId].p);
                }
            }

            // Insert new vertex in the working simplex
            vec4 vertex(verts[vId].p, f);
            if(vertex.w > simplex[3].w)
                swap(simplex[3], vertex);
            if(vertex.w > simplex[2].w)
                swap(simplex[2], vertex);
            if(vertex.w > simplex[1].w)
                swap(simplex[1], vertex);
            if(vertex.w > simplex[0].w)
                swap(simplex[0], vertex);


            if( (simplex[3].w - simplex[1].w) < VALUE_CONVERGENCE )
            {
                terminated = true;
                break;
            }
        }

        if( terminated || (cycle >= SECURITY_CYCLE_COUNT && reset) )
        {
            break;
        }
        else
        {
            simplex[0] = vo - vec4(nodeShift, 0, 0, 0);
            simplex[1] = vo - vec4(0, nodeShift, 0, 0);
            simplex[2] = vo - vec4(0, 0, nodeShift, 0);
            simplex[3] = vo;
            reset = true;
            cycle = 0;
        }
    }

    verts[vId].p = vec3(simplex[3]);
}

__global__ void smoothMultiElemNMVerticesCudaMain()
{
    uint vId = getInvocationVertexId();

    if(isSmoothableVertex(vId))
    {
        multiElemNMSmoothVert(vId);
    }
}

// __device__ smoothVertFct multiElemNMSmoothVertPtr = multiElemNMSmoothVert;


// CUDA Drivers
void setupCudaIndependentDispatch(const NodeGroups::GpuDispatch& dispatch);

void installCudaMultiElemNMSmoother(
        float h_valueConvergence,
        int h_securityCycleCount,
        float h_localSizeToNodeShift,
        float h_alpha,
        float h_beta,
        float h_gamma,
        float h_delta)
{
//    smoothVertFct d_smoothVert = nullptr;
//    hipMemcpyFromSymbol(&d_smoothVert, HIP_SYMBOL(multiElemNMSmoothVertPtr), sizeof(smoothVertFct));
//    hipMemcpyToSymbol(HIP_SYMBOL(smoothVert), &d_smoothVert, sizeof(smoothVertFct));

    hipMemcpyToSymbol(HIP_SYMBOL(VALUE_CONVERGENCE), &h_valueConvergence, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(SECURITY_CYCLE_COUNT), &h_securityCycleCount, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(LOCALE_SIZE_TO_NODE_SHIFT), &h_localSizeToNodeShift, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(ALPHA), &h_alpha, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(BETA),  &h_beta,  sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(GAMMA), &h_gamma, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(DELTA), &h_delta, sizeof(float));


    if(verboseCuda)
        printf("I -> CUDA \tMulti Elem Nelder Mead smoother installed\n");
}

void smoothCudaMultiElemNMVertices(
        const NodeGroups::GpuDispatch& dispatch)
{
    setupCudaIndependentDispatch(dispatch);

    dim3 blockDim(dispatch.workgroupSize.x,
                  dispatch.workgroupSize.y,
                  dispatch.workgroupSize.z);
    dim3 blockCount(dispatch.workgroupCount.x,
                    dispatch.workgroupCount.y,
                    dispatch.workgroupCount.z);

    cudaCheckErrors("CUDA error before vertices smoothing");
    smoothMultiElemNMVerticesCudaMain<<<blockCount, blockDim>>>();
    cudaCheckErrors("CUDA error during vertices smoothing");

    hipDeviceSynchronize();
}
