#include "hip/hip_runtime.h"
#include "Base.cuh"

#include <iostream>

#include <DataStructures/NodeGroups.h>

#define POSITION_THREAD_COUNT uint(8)
#define ELEMENT_THREAD_COUNT uint(32)
#define ELEMENT_SLOT_COUNT uint(96)
#define SLOT_PER_THREAD (ELEMENT_SLOT_COUNT / ELEMENT_THREAD_COUNT)

#define GRAD_SAMP_COUNT uint(6)
#define LINE_SAMP_COUNT uint(8)

__constant__ int PGDSecurityCycleCount;
__constant__ float PGDLocalSizeToNodeShift;

struct PatchElem
{
    __device__ PatchElem() {}

    uint type;
    uint n;

    Tet tet;
    Pri pri;
    Hex hex;
    vec3 p[HEX_VERTEX_COUNT];
};


// Smoothing Helper
__device__ float computeLocalElementSize(uint vId);
__device__ void accumulatePatchQuality(
        double& patchQuality,
        double& patchWeight,
        double elemQuality);
__device__ float finalizePatchQuality(
        double patchQuality,
        double patchWeight);

__shared__ float nodeShift;
__shared__ float3 lineShift;
__shared__ extern PatchElem patchElems[];
__shared__ float elemQual[POSITION_THREAD_COUNT][ELEMENT_SLOT_COUNT];
__shared__ float patchQual[POSITION_THREAD_COUNT];


// ENTRY POINT //
__device__ void patchGradDsntSmoothVert(uint vId)
{
    const vec3 GRAD_SAMPS[GRAD_SAMP_COUNT] = {
        vec3(-1, 0, 0), vec3( 1, 0, 0), vec3(0, -1, 0),
        vec3(0,  1, 0), vec3(0, 0, -1), vec3(0, 0,  1)
    };

    const float LINE_SAMPS[LINE_SAMP_COUNT] = {
        -0.25, 0.00, 0.25, 0.50,
         0.75, 1.00, 1.25, 1.50
    };

    uint pId = threadIdx.x;
    uint eId = threadIdx.y;

    Topo topo = topos[vId];
    uint neigElemCount = topo.neigElemCount;
    uint eBeg = (eId * neigElemCount) / ELEMENT_THREAD_COUNT;
    uint eEnd = ((eId+1) * neigElemCount) / ELEMENT_THREAD_COUNT;

    if(pId == 0)
    {
        for(uint e = eBeg; e < eEnd; ++e)
        {
            NeigElem elem = neigElems[topo.neigElemBase + e];
            patchElems[e].type = elem.type;
            patchElems[e].n = 0;

            switch(patchElems[e].type)
            {
            case TET_ELEMENT_TYPE :
                patchElems[e].tet = tets[elem.id];
                patchElems[e].p[0] = verts[patchElems[e].tet.v[0]].p;
                patchElems[e].p[1] = verts[patchElems[e].tet.v[1]].p;
                patchElems[e].p[2] = verts[patchElems[e].tet.v[2]].p;
                patchElems[e].p[3] = verts[patchElems[e].tet.v[3]].p;

                if(patchElems[e].tet.v[1] == vId) patchElems[e].n = 1;
                else if(patchElems[e].tet.v[2] == vId) patchElems[e].n = 2;
                else if(patchElems[e].tet.v[3] == vId) patchElems[e].n = 3;
                break;

            case PRI_ELEMENT_TYPE :
                patchElems[e].pri = pris[elem.id];
                patchElems[e].p[0] = verts[patchElems[e].pri.v[0]].p;
                patchElems[e].p[1] = verts[patchElems[e].pri.v[1]].p;
                patchElems[e].p[2] = verts[patchElems[e].pri.v[2]].p;
                patchElems[e].p[3] = verts[patchElems[e].pri.v[3]].p;
                patchElems[e].p[4] = verts[patchElems[e].pri.v[4]].p;
                patchElems[e].p[5] = verts[patchElems[e].pri.v[5]].p;

                if(patchElems[e].pri.v[1] == vId) patchElems[e].n = 1;
                else if(patchElems[e].pri.v[2] == vId) patchElems[e].n = 2;
                else if(patchElems[e].pri.v[3] == vId) patchElems[e].n = 3;
                else if(patchElems[e].pri.v[4] == vId) patchElems[e].n = 4;
                else if(patchElems[e].pri.v[5] == vId) patchElems[e].n = 5;
                break;

            case HEX_ELEMENT_TYPE :
                patchElems[e].hex = hexs[elem.id];
                patchElems[e].p[0] = verts[patchElems[e].hex.v[0]].p;
                patchElems[e].p[1] = verts[patchElems[e].hex.v[1]].p;
                patchElems[e].p[2] = verts[patchElems[e].hex.v[2]].p;
                patchElems[e].p[3] = verts[patchElems[e].hex.v[3]].p;
                patchElems[e].p[4] = verts[patchElems[e].hex.v[4]].p;
                patchElems[e].p[5] = verts[patchElems[e].hex.v[5]].p;
                patchElems[e].p[6] = verts[patchElems[e].hex.v[6]].p;
                patchElems[e].p[7] = verts[patchElems[e].hex.v[7]].p;

                if(patchElems[e].hex.v[1] == vId) patchElems[e].n = 1;
                else if(patchElems[e].hex.v[2] == vId) patchElems[e].n = 2;
                else if(patchElems[e].hex.v[3] == vId) patchElems[e].n = 3;
                else if(patchElems[e].hex.v[4] == vId) patchElems[e].n = 4;
                else if(patchElems[e].hex.v[5] == vId) patchElems[e].n = 5;
                else if(patchElems[e].hex.v[6] == vId) patchElems[e].n = 6;
                else if(patchElems[e].hex.v[7] == vId) patchElems[e].n = 7;
                break;
            }
        }
    }

    if(pId == 0 && eId == 0)
    {
        // Compute local element size
        float localSize = computeLocalElementSize(vId);

        // Initialize node shift distance
        nodeShift = localSize * PGDLocalSizeToNodeShift;
    }

    __syncthreads();

    float originalNodeShift = nodeShift;
    for(int c=0; c < PGDSecurityCycleCount; ++c)
    {
        vec3 pos = verts[vId].p;

        for(uint e = eBeg; e < eEnd; ++e)
        {
            vec3 vertPos[HEX_VERTEX_COUNT] = {
                patchElems[e].p[0],
                patchElems[e].p[1],
                patchElems[e].p[2],
                patchElems[e].p[3],
                patchElems[e].p[4],
                patchElems[e].p[5],
                patchElems[e].p[6],
                patchElems[e].p[7]
            };

            vertPos[patchElems[e].n] = pos + vec3(GRAD_SAMPS[pId]) * nodeShift;

            switch(patchElems[e].type)
            {
            case TET_ELEMENT_TYPE :
                elemQual[pId][e] = (*tetQualityImpl)(vertPos, patchElems[e].tet);
                break;
            case PRI_ELEMENT_TYPE :
                elemQual[pId][e] = (*priQualityImpl)(vertPos, patchElems[e].pri);
                break;
            case HEX_ELEMENT_TYPE :
                elemQual[pId][e] = (*hexQualityImpl)(vertPos, patchElems[e].hex);
                break;
            }
        }

        __syncthreads();

        if(eId == 0 && pId < GRAD_SAMP_COUNT)
        {
            double patchWeight = 0.0;
            double patchQuality = 0.0;
            for(uint e = 0; e < neigElemCount; ++e)
                accumulatePatchQuality(
                    patchQuality, patchWeight,
                    double(elemQual[pId][e]));

            patchQual[pId] = finalizePatchQuality(patchQuality, patchWeight);
        }

        __syncthreads();

        if(eId == 0 && pId == 0)
        {
            vec3 gradQ = vec3(
                patchQual[1] - patchQual[0],
                patchQual[3] - patchQual[2],
                patchQual[5] - patchQual[4]);
            float gradQNorm = length(gradQ);

            if(gradQNorm != 0)
            {
                lineShift = toFloat3(gradQ * (nodeShift / gradQNorm));
            }
            else
            {
                lineShift = make_float3(0, 0, 0);
            }
        }

        __syncthreads();

        if(lineShift.x == 0 && lineShift.y == 0 && lineShift.z == 0)
            break;

        for(uint e = eBeg; e < eEnd; ++e)
        {
            vec3 vertPos[HEX_VERTEX_COUNT] = {
                patchElems[e].p[0],
                patchElems[e].p[1],
                patchElems[e].p[2],
                patchElems[e].p[3],
                patchElems[e].p[4],
                patchElems[e].p[5],
                patchElems[e].p[6],
                patchElems[e].p[7]
            };

            vertPos[patchElems[e].n] = pos + toVec3(lineShift) * LINE_SAMPS[pId];
            switch(patchElems[e].type)
            {
            case TET_ELEMENT_TYPE :
                elemQual[pId][e] = (*tetQualityImpl)(vertPos, patchElems[e].tet);
                break;
            case PRI_ELEMENT_TYPE :
                elemQual[pId][e] = (*priQualityImpl)(vertPos, patchElems[e].pri);
                break;
            case HEX_ELEMENT_TYPE :
                elemQual[pId][e] = (*hexQualityImpl)(vertPos, patchElems[e].hex);
                break;
            }
        }

        __syncthreads();

        if(eId == 0)
        {
            double patchWeight = 0.0;
            double patchQuality = 0.0;
            for(uint e = 0; e < neigElemCount; ++e)
                accumulatePatchQuality(
                    patchQuality, patchWeight,
                    double(elemQual[pId][e]));

            patchQual[pId] = finalizePatchQuality(patchQuality, patchWeight);
        }

        __syncthreads();

        if(eId == 0 && pId == 0)
        {
            uint bestProposition = 0;
            float bestQualityMean = patchQual[0];
            for(uint p=1; p < LINE_SAMP_COUNT; ++p)
            {
                if(patchQual[p] > bestQualityMean)
                {
                    bestQualityMean = patchQual[p];
                    bestProposition = p;
                }
            }

            // Update vertex's position
            verts[vId].p = pos + toVec3(lineShift) * LINE_SAMPS[bestProposition];

            // Scale node shift and stop if it is too small
            nodeShift *= abs(LINE_SAMPS[bestProposition]);
        }

        __syncthreads();

        if(nodeShift < originalNodeShift / 10.0)
            break;
    }
}

__global__ void smoothPatchGradDsntVerticesCudaMain()
{
    if(blockIdx.x < GroupSize)
    {
        uint idx = GroupBase + blockIdx.x;
        uint vId = groupMembers[idx];
        patchGradDsntSmoothVert(vId);
    }
}

//__device__ smoothVertFct patchGradDsntSmoothVertPtr = patchGradDsntSmoothVert;



// CUDA Drivers
void setupCudaIndependentDispatch(const NodeGroups::GpuDispatch& dispatch);

void installCudaPatchGradDsntSmoother(
        int h_securityCycleCount,
        float h_localSizeToNodeShift)
{
    // Main function is directly calling patchGradDsntSmoothVert

//    smoothVertFct d_smoothVert = nullptr;
//    hipMemcpyFromSymbol(&d_smoothVert, HIP_SYMBOL(patchGradDsntSmoothVertPtr), sizeof(smoothVertFct));
//    hipMemcpyToSymbol(HIP_SYMBOL(smoothVert), &d_smoothVert, sizeof(smoothVertFct));

    hipMemcpyToSymbol(HIP_SYMBOL(PGDSecurityCycleCount), &h_securityCycleCount, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(PGDLocalSizeToNodeShift), &h_localSizeToNodeShift, sizeof(float));


    if(verboseCuda)
        printf("I -> CUDA \tPatch Gradient Decsent smoother installed\n");
}

void smoothCudaPatchGradDsntVertices(
        const NodeGroups::GpuDispatch& dispatch)
{
    setupCudaIndependentDispatch(dispatch);

    dim3 blockDim(POSITION_THREAD_COUNT, ELEMENT_THREAD_COUNT);
    size_t sharedDim = sizeof(PatchElem) * ELEMENT_SLOT_COUNT;
    //std::cout << "Requested shared memory size: " << sharedDim/1000.0 << "kB" << std::endl;

    cudaCheckErrors("CUDA error before vertices smoothing");
    smoothPatchGradDsntVerticesCudaMain<<<dispatch.gpuBufferSize, blockDim, sharedDim>>>();
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA error during vertices smoothing");
}
