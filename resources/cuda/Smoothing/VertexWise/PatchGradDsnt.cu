#include "hip/hip_runtime.h"
#include "Base.cuh"

#include <iostream>

#include <DataStructures/NodeGroups.h>

#define POSITION_THREAD_COUNT uint(8)
#define ELEMENT_THREAD_COUNT uint(8)

#define ELEMENT_PER_THREAD_COUNT uint(96 / ELEMENT_THREAD_COUNT)

#define GRAD_SAMP_COUNT uint(6)
#define LINE_SAMP_COUNT uint(8)

#define MIN_MAX 2147483647


namespace pgd
{
    __constant__ int SECURITY_CYCLE_COUNT;
    __constant__ float LOCAL_SIZE_TO_NODE_SHIFT;

    __shared__ float nodeShift;
    __shared__ int patchMin[POSITION_THREAD_COUNT];
    __shared__ float patchMean[POSITION_THREAD_COUNT];
    __shared__ float patchQual[POSITION_THREAD_COUNT];
}

using namespace pgd;


// Smoothing Helper
__device__ float computeLocalElementSize(uint vId);


// ENTRY POINT //
__device__ void patchGradDsntSmoothVert(uint vId)
{
    const vec3 GRAD_SAMPS[GRAD_SAMP_COUNT] = {
        vec3(-1, 0, 0), vec3( 1, 0, 0), vec3(0, -1, 0),
        vec3(0,  1, 0), vec3(0, 0, -1), vec3(0, 0,  1)
    };

    const float LINE_SAMPS[LINE_SAMP_COUNT] = {
        -0.25, 0.00, 0.25, 0.50,
         0.75, 1.00, 1.25, 1.50
    };

    uint pId = threadIdx.x;
    uint eId = threadIdx.y;

    Topo topo = topos[vId];
    uint neigBase = topo.neigElemBase;
    uint neigElemCount = topo.neigElemCount;
    uint eBeg = (eId * neigElemCount) / ELEMENT_THREAD_COUNT;
    uint eEnd = ((eId+1) * neigElemCount) / ELEMENT_THREAD_COUNT;
    uint nBeg = neigBase + eBeg;
    uint nEnd = neigBase + eEnd;

    PatchElem elems[ELEMENT_PER_THREAD_COUNT];
    for(uint e=0, ne = nBeg; ne < nEnd; ++e, ++ne)
    {
        NeigElem elem = neigElems[ne];
        elems[e].type = elem.type;
        elems[e].n = elem.vId;

        switch(elems[e].type)
        {
        case TET_ELEMENT_TYPE :
            elems[e].tet = tets[elem.id];
            elems[e].p[0] = verts[elems[e].tet.v[0]].p;
            elems[e].p[1] = verts[elems[e].tet.v[1]].p;
            elems[e].p[2] = verts[elems[e].tet.v[2]].p;
            elems[e].p[3] = verts[elems[e].tet.v[3]].p;
            break;

        case PRI_ELEMENT_TYPE :
            elems[e].pri = pris[elem.id];
            elems[e].p[0] = verts[elems[e].pri.v[0]].p;
            elems[e].p[1] = verts[elems[e].pri.v[1]].p;
            elems[e].p[2] = verts[elems[e].pri.v[2]].p;
            elems[e].p[3] = verts[elems[e].pri.v[3]].p;
            elems[e].p[4] = verts[elems[e].pri.v[4]].p;
            elems[e].p[5] = verts[elems[e].pri.v[5]].p;
            break;

        case HEX_ELEMENT_TYPE :
            elems[e].hex = hexs[elem.id];
            elems[e].p[0] = verts[elems[e].hex.v[0]].p;
            elems[e].p[1] = verts[elems[e].hex.v[1]].p;
            elems[e].p[2] = verts[elems[e].hex.v[2]].p;
            elems[e].p[3] = verts[elems[e].hex.v[3]].p;
            elems[e].p[4] = verts[elems[e].hex.v[4]].p;
            elems[e].p[5] = verts[elems[e].hex.v[5]].p;
            elems[e].p[6] = verts[elems[e].hex.v[6]].p;
            elems[e].p[7] = verts[elems[e].hex.v[7]].p;
            break;
        }
    }

    if(eId == 0)
    {
        patchMin[pId] = MIN_MAX;
        patchMean[pId] = 0.0;
    }

    if(pId == 0 && eId == 0)
    {
        // Compute local element size
        float localSize = computeLocalElementSize(vId);

        // Initialize node shift distance
        nodeShift = localSize * LOCAL_SIZE_TO_NODE_SHIFT;
    }

    __syncthreads();


    float originalNodeShift = nodeShift;
    for(int c=0; c < SECURITY_CYCLE_COUNT; ++c)
    {
        vec3 pos = verts[vId].p;

        if(pId < GRAD_SAMP_COUNT)
        {
            vec3 gradSamp = pos + GRAD_SAMPS[pId] * nodeShift;

            for(uint e=0, id = eBeg; id < eEnd; ++e, ++id)
            {
                elems[e].p[elems[e].n] = gradSamp;

                float qual = 0.0;
                switch(elems[e].type)
                {
                case TET_ELEMENT_TYPE :
                    qual = (*tetQualityImpl)(elems[e].p, elems[e].tet);
                    break;
                case PRI_ELEMENT_TYPE :
                    qual = (*priQualityImpl)(elems[e].p, elems[e].pri);
                    break;
                case HEX_ELEMENT_TYPE :
                    qual = (*hexQualityImpl)(elems[e].p, elems[e].hex);
                    break;
                }

                atomicMin(&patchMin[pId], qual * MIN_MAX);
                atomicAdd(&patchMean[pId], 1.0 / qual);
            }
        }

        __syncthreads();


        if(eId == 0)
        {
            if(patchMin[pId] <= 0.0)
                patchQual[pId] = patchMin[pId] / float(MIN_MAX);
            else
                patchQual[pId] = neigElemCount / patchMean[pId];

            patchMin[pId] = MIN_MAX;
            patchMean[pId] = 0.0;
        }

        __syncthreads();


        vec3 gradQ = vec3(
            patchQual[1] - patchQual[0],
            patchQual[3] - patchQual[2],
            patchQual[5] - patchQual[4]);
        float gradQNorm = length(gradQ);

        vec3 lineShift;
        if(gradQNorm != 0)
            lineShift = gradQ * (nodeShift / gradQNorm);
        else
            break;


        vec3 lineSamp = pos + lineShift * LINE_SAMPS[pId];

        for(uint e=0, id = eBeg; id < eEnd; ++e, ++id)
        {
            elems[e].p[elems[e].n] = lineSamp;

            float qual = 0.0;
            switch(elems[e].type)
            {
            case TET_ELEMENT_TYPE :
                qual = (*tetQualityImpl)(elems[e].p, elems[e].tet);
                break;
            case PRI_ELEMENT_TYPE :
                qual = (*priQualityImpl)(elems[e].p, elems[e].pri);
                break;
            case HEX_ELEMENT_TYPE :
                qual = (*hexQualityImpl)(elems[e].p, elems[e].hex);
                break;
            }

            atomicMin(&patchMin[pId], qual * MIN_MAX);
            atomicAdd(&patchMean[pId], 1.0 / qual);
        }

        __syncthreads();


        if(eId == 0)
        {
            if(patchMin[pId] <= 0.0)
                patchQual[pId] = patchMin[pId] / float(MIN_MAX);
            else
                patchQual[pId] = neigElemCount / patchMean[pId];

            patchMin[pId] = MIN_MAX;
            patchMean[pId] = 0.0;
        }

        __syncthreads();


        if(eId == 0 && pId == 0)
        {
            uint bestProposition = 0;
            float bestQualityMean = patchQual[0];
            for(uint p=1; p < LINE_SAMP_COUNT; ++p)
            {
                if(patchQual[p] > bestQualityMean)
                {
                    bestQualityMean = patchQual[p];
                    bestProposition = p;
                }
            }

            // Update vertex's position
            verts[vId].p = pos + lineShift * LINE_SAMPS[bestProposition];

            // Scale node shift and stop if it is too small
            nodeShift *= abs(LINE_SAMPS[bestProposition]);
        }

        __syncthreads();


        if(nodeShift < originalNodeShift / 10.0)
            break;
    }
}


__global__ void smoothPatchGradDsntVerticesCudaMain()
{
    if(blockIdx.x < GroupSize)
    {
        uint idx = GroupBase + blockIdx.x;
        uint vId = groupMembers[idx];
        patchGradDsntSmoothVert(vId);
    }
}

//__device__ smoothVertFct patchGradDsntSmoothVertPtr = patchGradDsntSmoothVert;



// CUDA Drivers
void setupCudaIndependentDispatch(const NodeGroups::GpuDispatch& dispatch);

void installCudaPatchGradDsntSmoother(
        int h_securityCycleCount,
        float h_localSizeToNodeShift)
{
    // Main function is directly calling patchGradDsntSmoothVert

//    smoothVertFct d_smoothVert = nullptr;
//    hipMemcpyFromSymbol(&d_smoothVert, HIP_SYMBOL(patchGradDsntSmoothVertPtr), sizeof(smoothVertFct));
//    hipMemcpyToSymbol(HIP_SYMBOL(smoothVert), &d_smoothVert, sizeof(smoothVertFct));

    hipMemcpyToSymbol(HIP_SYMBOL(SECURITY_CYCLE_COUNT), &h_securityCycleCount, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(LOCAL_SIZE_TO_NODE_SHIFT), &h_localSizeToNodeShift, sizeof(float));

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(smoothPatchGradDsntVerticesCudaMain), hipFuncCachePreferL1);

    if(verboseCuda)
        printf("I -> CUDA \tPatch Gradient Decsent smoother installed\n");
}

void smoothCudaPatchGradDsntVertices(
        const NodeGroups::GpuDispatch& dispatch)
{
    setupCudaIndependentDispatch(dispatch);

    dim3 blockDim(POSITION_THREAD_COUNT, ELEMENT_THREAD_COUNT);

    cudaCheckErrors("CUDA error before vertices smoothing");
    smoothPatchGradDsntVerticesCudaMain<<<dispatch.workgroupCount, blockDim>>>();
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA error during vertices smoothing");
}
