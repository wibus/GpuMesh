#include "Base.cuh"


__device__  float SLMoveCoeff = 0.35;


// ENTRY POINT //
__device__ void springLaplaceSmoothVert(uint vId)
{
    vec3 patchCenter = computeVertexEquilibrium(vId);
    verts[vId].p = mix(verts[vId].p, patchCenter, SLMoveCoeff);
}

__device__ smoothVertFct springLaplaceSmoothVertPtr = springLaplaceSmoothVert;


// CUDA Drivers
void installCudaSpringLaplaceSmoother(float moveCoeff)
{
    hipMemcpyToSymbol(HIP_SYMBOL(SLMoveCoeff), &moveCoeff, sizeof(float));

    smoothVertFct d_smoothVert = nullptr;
    hipMemcpyFromSymbol(&d_smoothVert, HIP_SYMBOL(springLaplaceSmoothVertPtr), sizeof(smoothVertFct));
    hipMemcpyToSymbol(HIP_SYMBOL(smoothVert), &d_smoothVert, sizeof(smoothVertFct));


    if(verboseCuda)
        printf("I -> CUDA \tSpring Laplace smoother installed\n");
}
