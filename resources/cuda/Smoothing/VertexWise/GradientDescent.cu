#include "hip/hip_runtime.h"
#include "Base.cuh"

__constant__ int GDSecurityCycleCount;
__constant__ float GDLocalSizeToNodeShift;


// Smoothing Helper
__device__ float computeLocalElementSize(uint vId);
__device__ float patchQuality(uint vId);


// ENTRY POINT //
__device__ void gradientDescentSmoothVert(uint vId)
{
    // Compute local element size
    float localSize = computeLocalElementSize(vId);

    // Initialize node shift distance
    float nodeShift = localSize * GDLocalSizeToNodeShift;
    float originalNodeShift = nodeShift;

    for(int c=0; c < GDSecurityCycleCount; ++c)
    {
        // Define patch quality gradient samples
        vec3 pos = vec3(verts[vId].p);
        const uint GRADIENT_SAMPLE_COUNT = 6;
        float sampleQualities[GRADIENT_SAMPLE_COUNT] =
            {1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
        vec3 gradSamples[GRADIENT_SAMPLE_COUNT] = {
            pos + vec3(-nodeShift, 0.0,   0.0),
            pos + vec3( nodeShift, 0.0,   0.0),
            pos + vec3( 0.0,  -nodeShift, 0.0),
            pos + vec3( 0.0,   nodeShift, 0.0),
            pos + vec3( 0.0,   0.0,  -nodeShift),
            pos + vec3( 0.0,   0.0,   nodeShift)
        };

        Topo topo = topos[vId];
        if(topo.type > 0)
        {
            for(uint p=0; p < GRADIENT_SAMPLE_COUNT; ++p)
                gradSamples[p] = snapToBoundary(
                    topo.type, gradSamples[p]);
        }

        for(uint p=0; p < GRADIENT_SAMPLE_COUNT; ++p)
        {
            // Quality evaluation functions will use this updated position
            // to compute element shape measures.
            verts[vId].p = vec4(gradSamples[p], 0.0);

            // Compute patch quality
            sampleQualities[p] = patchQuality(vId);
        }
        verts[vId].p = vec4(pos, 0.0);

        vec3 gradQ = vec3(
            sampleQualities[1] - sampleQualities[0],
            sampleQualities[3] - sampleQualities[2],
            sampleQualities[5] - sampleQualities[4]);
        float gradQNorm = length(gradQ);

        if(gradQNorm == 0)
            break;


        const uint PROPOSITION_COUNT = 7;
        const float OFFSETS[PROPOSITION_COUNT] = {
            -0.25,
             0.00,
             0.25,
             0.50,
             0.75,
             1.00,
             1.25
        };

        vec3 shift = gradQ * (nodeShift / gradQNorm);
        vec3 propositions[PROPOSITION_COUNT] = {
            pos + shift * OFFSETS[0],
            pos + shift * OFFSETS[1],
            pos + shift * OFFSETS[2],
            pos + shift * OFFSETS[3],
            pos + shift * OFFSETS[4],
            pos + shift * OFFSETS[5],
            pos + shift * OFFSETS[6]
        };

        if(topo.type > 0)
        {
            for(uint p=0; p < PROPOSITION_COUNT; ++p)
                propositions[p] = snapToBoundary(
                    topo.type, propositions[p]);
        }

        uint bestProposition = 0;
        float bestQualityMean = -1.0/0.0; // -Inf
        for(uint p=0; p < PROPOSITION_COUNT; ++p)
        {
            // Quality evaluation functions will use this updated position
            // to compute element shape measures.
            verts[vId].p = vec4(propositions[p], 0.0);

            // Compute patch quality
            float pq = patchQuality(vId);

            if(pq > bestQualityMean)
            {
                bestQualityMean = pq;
                bestProposition = p;
            }
        }


        // Update vertex's position
        verts[vId].p = vec4(propositions[bestProposition], 0.0);

        // Scale node shift and stop if it is too small
        nodeShift *= abs(OFFSETS[bestProposition]);
        if(nodeShift < originalNodeShift / 10.0)
            break;
    }
}

__device__ smoothVertFct gradientDescentSmoothVertPtr = gradientDescentSmoothVert;


// CUDA Drivers
void installCudaGradientDescentSmoother(
        int h_securityCycleCount,
        float h_localSizeToNodeShift)
{
    smoothVertFct d_smoothVert = nullptr;
    hipMemcpyFromSymbol(&d_smoothVert, HIP_SYMBOL(gradientDescentSmoothVertPtr), sizeof(smoothVertFct));
    hipMemcpyToSymbol(HIP_SYMBOL(smoothVert), &d_smoothVert, sizeof(smoothVertFct));

    // TODO wbussiere 2016-04-04 : Pass security cycle count and
    //  local size to node shift from Smoother

    hipMemcpyToSymbol(HIP_SYMBOL(GDSecurityCycleCount), &h_securityCycleCount, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(GDLocalSizeToNodeShift), &h_localSizeToNodeShift, sizeof(float));

    printf("I -> CUDA \tGradient Descent smoother installed\n");
}
