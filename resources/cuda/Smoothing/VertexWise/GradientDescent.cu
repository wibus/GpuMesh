#include "hip/hip_runtime.h"
#include "Base.cuh"

namespace gd
{
    __constant__ int SECURITY_CYCLE_COUNT;
    __constant__ float LOCAL_SIZE_TONODE_SHIFT;
}

using namespace gd;


// Smoothing Helper
__device__ float computeLocalElementSize(uint vId);
__device__ float patchQuality(uint vId);


// ENTRY POINT //
__device__ void gradientDescentSmoothVert(uint vId)
{
    // Compute local element size
    float localSize = computeLocalElementSize(vId);

    // Initialize node shift distance
    float nodeShift = localSize * LOCAL_SIZE_TONODE_SHIFT;
    float originalNodeShift = nodeShift;

    for(int c=0; c < SECURITY_CYCLE_COUNT; ++c)
    {
        // Define patch quality gradient samples
        vec3 pos = verts[vId].p;
        const uint GRADIENT_SAMPLE_COUNT = 6;
        float sampleQualities[GRADIENT_SAMPLE_COUNT] =
            {1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
        vec3 gradSamples[GRADIENT_SAMPLE_COUNT] = {
            pos + vec3(-nodeShift, 0.0,   0.0),
            pos + vec3( nodeShift, 0.0,   0.0),
            pos + vec3( 0.0,  -nodeShift, 0.0),
            pos + vec3( 0.0,   nodeShift, 0.0),
            pos + vec3( 0.0,   0.0,  -nodeShift),
            pos + vec3( 0.0,   0.0,   nodeShift)
        };

        for(uint p=0; p < GRADIENT_SAMPLE_COUNT; ++p)
        {
            // Quality evaluation functions will use this updated position
            // to compute element shape measures.
            verts[vId].p = gradSamples[p];

            // Compute patch quality
            sampleQualities[p] = patchQuality(vId);
        }
        verts[vId].p = pos, 0.0;

        vec3 gradQ = vec3(
            sampleQualities[1] - sampleQualities[0],
            sampleQualities[3] - sampleQualities[2],
            sampleQualities[5] - sampleQualities[4]);
        float gradQNorm = length(gradQ);

        if(gradQNorm == 0)
            break;


        const uint PROPOSITION_COUNT = 8;
        const float OFFSETS[PROPOSITION_COUNT] = {
            -0.25, 0.00, 0.10, 0.20,
             0.40, 0.80, 1.20, 1.60
        };

        vec3 shift = gradQ * (nodeShift / gradQNorm);
        vec3 propositions[PROPOSITION_COUNT] = {
            pos + shift * OFFSETS[0],
            pos + shift * OFFSETS[1],
            pos + shift * OFFSETS[2],
            pos + shift * OFFSETS[3],
            pos + shift * OFFSETS[4],
            pos + shift * OFFSETS[5],
            pos + shift * OFFSETS[6],
            pos + shift * OFFSETS[7]
        };

        uint bestProposition = 0;
        float bestQualityMean = -1.0/0.0; // -Inf
        for(uint p=0; p < PROPOSITION_COUNT; ++p)
        {
            // Quality evaluation functions will use this updated position
            // to compute element shape measures.
            verts[vId].p = propositions[p];

            // Compute patch quality
            float pq = patchQuality(vId);

            if(pq > bestQualityMean)
            {
                bestQualityMean = pq;
                bestProposition = p;
            }
        }


        // Update vertex's position
        verts[vId].p = propositions[bestProposition];

        // Scale node shift and stop if it is too small
        nodeShift *= abs(OFFSETS[bestProposition]);
        if(nodeShift < originalNodeShift / 10.0)
            break;
    }
}

__device__ smoothVertFct gradientDescentSmoothVertPtr = gradientDescentSmoothVert;


// CUDA Drivers
void installCudaGradientDescentSmoother(
        int h_securityCycleCount,
        float h_localSizeToNodeShift)
{
    smoothVertFct d_smoothVert = nullptr;
    hipMemcpyFromSymbol(&d_smoothVert, HIP_SYMBOL(gradientDescentSmoothVertPtr), sizeof(smoothVertFct));
    hipMemcpyToSymbol(HIP_SYMBOL(smoothVert), &d_smoothVert, sizeof(smoothVertFct));

    hipMemcpyToSymbol(HIP_SYMBOL(SECURITY_CYCLE_COUNT), &h_securityCycleCount, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(LOCAL_SIZE_TONODE_SHIFT), &h_localSizeToNodeShift, sizeof(float));

    if(verboseCuda)
        printf("I -> CUDA \tGradient Descent smoother installed\n");
}
