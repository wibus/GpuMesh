#include "Base.cuh"

__constant__ float NMValueConvergence;
__constant__ int NMSecurityCycleCount;
__constant__ float NMLocalSizeToNodeShift;
__constant__ float NMAlpha;
__constant__ float NMBeta;
__constant__ float NMGamma;
__constant__ float NMDelta;


// Smoothing Helper
__device__ float computeLocalElementSize(uint vId);
__device__ float patchQuality(uint vId);
__device__ void swap(vec4& v1, vec4& v2)
{
    glm::dvec4 tmp = v1;
    v1 = v2;
    v2 = tmp;
}

// ENTRY POINT //
__device__ void nelderMeadSmoothVert(uint vId)
{
    // Compute local element size
    float localSize = computeLocalElementSize(vId);

    // Initialize node shift distance
    float nodeShift = localSize * NMLocalSizeToNodeShift;


    Topo topo = topos[vId];
    vec3 pos = vec3(verts[vId].p);
    vec4 vo(pos, patchQuality(vId));

    vec4 simplex[TET_VERTEX_COUNT] = {
        vec4(pos + vec3(nodeShift, 0, 0), 0),
        vec4(pos + vec3(0, nodeShift, 0), 0),
        vec4(pos + vec3(0, 0, nodeShift), 0),
        vo
    };

    int cycle = 0;
    bool reset = false;
    bool terminated = false;
    while(!terminated)
    {
        for(uint p=0; p < TET_VERTEX_COUNT-1; ++p)
        {
            // Since 'pos' is a reference on vertex's position
            // modifing its value here should be seen by the evaluator
            if(topo.type > 0)
                verts[vId].p = vec4(snapToBoundary(topo.type, vec3(simplex[p])), 0);
            else
                verts[vId].p = simplex[p];

            // Compute patch quality
            simplex[p] = vec4(vec3(verts[vId].p), patchQuality(vId));
        }

        // Mini bubble sort
        if(simplex[0].w > simplex[1].w)
            swap(simplex[0], simplex[1]);
        if(simplex[1].w > simplex[2].w)
            swap(simplex[1], simplex[2]);
        if(simplex[2].w > simplex[3].w)
            swap(simplex[2], simplex[3]);
        if(simplex[0].w > simplex[1].w)
            swap(simplex[0], simplex[1]);
        if(simplex[1].w > simplex[2].w)
            swap(simplex[1], simplex[2]);
        if(simplex[0].w > simplex[1].w)
            swap(simplex[0], simplex[1]);


        for(; cycle < NMSecurityCycleCount; ++cycle)
        {
            // Centroid
            vec3 c = 1/3.0f * (
                vec3(simplex[1]) +
                vec3(simplex[2]) +
                vec3(simplex[3]));

            float f = 0.0;

            // Reflect
            verts[vId].p = vec4(c + NMAlpha*(c - vec3(simplex[0])), 0);
            if(topo.type > 0) verts[vId].p = vec4(snapToBoundary(topo.type, vec3(verts[vId].p)), 0);
            float fr = f = patchQuality(vId);

            vec4 xr = verts[vId].p;

            // Expand
            if(simplex[3].w < fr)
            {
                verts[vId].p = vec4(c + NMGamma*(vec3(verts[vId].p) - c), 0);
                if(topo.type > 0) verts[vId].p = vec4(snapToBoundary(topo.type, vec3(verts[vId].p)), 0);
                float fe = f = patchQuality(vId);

                if(fe <= fr)
                {
                    verts[vId].p = xr;
                    f = fr;
                }
            }
            // Contract
            else if(simplex[1].w >= fr)
            {
                // Outside
                if(fr > simplex[0].w)
                {
                    verts[vId].p = vec4(c + NMBeta*(vec3(xr) - c), 0);
                    if(topo.type > 0) verts[vId].p = vec4(snapToBoundary(topo.type, vec3(verts[vId].p)), 0);
                    f = patchQuality(vId);
                }
                // Inside
                else
                {
                    verts[vId].p = vec4(c + NMBeta*(vec3(simplex[0]) - c), 0);
                    if(topo.type > 0) verts[vId].p = vec4(snapToBoundary(topo.type, vec3(verts[vId].p)), 0);
                    f = patchQuality(vId);
                }
            }

            // Insert new vertex in the working simplex
            vec4 vertex(vec3(verts[vId].p), f);
            if(vertex.w > simplex[3].w)
                swap(simplex[3], vertex);
            if(vertex.w > simplex[2].w)
                swap(simplex[2], vertex);
            if(vertex.w > simplex[1].w)
                swap(simplex[1], vertex);
            if(vertex.w > simplex[0].w)
                swap(simplex[0], vertex);


            if( (simplex[3].w - simplex[1].w) < NMValueConvergence )
            {
                terminated = true;
                break;
            }
        }

        if( terminated || (cycle >= NMSecurityCycleCount && reset) )
        {
            break;
        }
        else
        {
            simplex[0] = vo - vec4(nodeShift, 0, 0, 0);
            simplex[1] = vo - vec4(0, nodeShift, 0, 0);
            simplex[2] = vo - vec4(0, 0, nodeShift, 0);
            simplex[3] = vo;
            reset = true;
            cycle = 0;
        }
    }

    if(topo.type > 0)
        verts[vId].p = vec4(snapToBoundary(topo.type, vec3(simplex[3])), 0);
    else
        verts[vId].p = simplex[3];
}

__device__ smoothVertFct nelderMeadSmoothVertPtr = nelderMeadSmoothVert;


// CUDA Drivers
void installCudaNelderMeadSmoother(
        float h_valueConvergence,
        int h_securityCycleCount,
        float h_localSizeToNodeShift,
        float h_alpha,
        float h_beta,
        float h_gamma,
        float h_delta)
{
    smoothVertFct d_smoothVert = nullptr;
    hipMemcpyFromSymbol(&d_smoothVert, HIP_SYMBOL(nelderMeadSmoothVertPtr), sizeof(smoothVertFct));
    hipMemcpyToSymbol(HIP_SYMBOL(smoothVert), &d_smoothVert, sizeof(smoothVertFct));

    // TODO wbussiere 2016-04-04 : Pass security cycle count and
    //  local size to node shift from Smoother
    hipMemcpyToSymbol(HIP_SYMBOL(NMValueConvergence), &h_valueConvergence, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(NMSecurityCycleCount), &h_securityCycleCount, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(NMLocalSizeToNodeShift), &h_localSizeToNodeShift, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(NMAlpha), &h_alpha, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(NMBeta),  &h_beta,  sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(NMGamma), &h_gamma, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(NMDelta), &h_delta, sizeof(float));


    if(verboseCuda)
        printf("I -> CUDA \tNelder Mead smoother installed\n");
}
