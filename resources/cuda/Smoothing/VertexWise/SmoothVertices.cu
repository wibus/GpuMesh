#include "hip/hip_runtime.h"
#include "Base.cuh"
#include <Smoothers/AbstractSmoother.h>


__device__ smoothVertFct smoothVert = nullptr;

__device__  float MoveCoeff = 0.35;


// Smoothing Helper
__device__ uint getInvocationVertexId();
__device__ bool isSmoothableVertex(uint vId);


__global__ void smoothVerticesCudaMain()
{
    uint vId = getInvocationVertexId();

    if(isSmoothableVertex(vId))
    {
        smoothVert(vId);
    }
}


// CUDA Drivers
void setupCudaIndependentDispatch(const IndependentDispatch& dispatch);

void smoothCudaVertices(
        const IndependentDispatch& dispatch,
        size_t workgroupSize,
        float moveCoeff)
{
    setupCudaIndependentDispatch(dispatch);
    hipMemcpyToSymbol(HIP_SYMBOL(MoveCoeff), &moveCoeff, sizeof(float));

    cudaCheckErrors("CUDA error before vertices smoothing");
    smoothVerticesCudaMain<<<dispatch.workgroupCount, workgroupSize>>>();
    cudaCheckErrors("CUDA error in vertices smoothing");

    hipDeviceSynchronize();
}
