#include "hip/hip_runtime.h"
#include "Base.cuh"
#include <DataStructures/NodeGroups.h>
#include <Smoothers/AbstractSmoother.h>


__device__ smoothVertFct smoothVert = nullptr;

__device__  float MoveCoeff = 0.35;


// Smoothing Helper
__device__ uint getInvocationVertexId();
__device__ bool isSmoothableVertex(uint vId);


__global__ void smoothVerticesCudaMain()
{
    uint vId = getInvocationVertexId();

    if(isSmoothableVertex(vId))
    {
        smoothVert(vId);
    }
}


// CUDA Drivers
void setupCudaIndependentDispatch(const NodeGroups::GpuDispatch& dispatch);

void smoothCudaVertices(
        const NodeGroups::GpuDispatch& dispatch,
        size_t workgroupSize,
        float moveCoeff)
{
    setupCudaIndependentDispatch(dispatch);
    hipMemcpyToSymbol(HIP_SYMBOL(MoveCoeff), &moveCoeff, sizeof(float));

    cudaCheckErrors("CUDA error before vertices smoothing");
    smoothVerticesCudaMain<<<dispatch.workgroupCount, workgroupSize>>>();
    cudaCheckErrors("CUDA error during vertices smoothing");

    hipDeviceSynchronize();
}
