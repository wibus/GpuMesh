#include "Base.cuh"

#define PIPE_CYLINDER_FACE_ID 1
#define PIPE_YNEG_DISK_FACE_ID 2
#define PIPE_YPOS_DISK_FACE_ID 3
#define PIPE_YNEG_CIRCLE_EDGE_ID 4
#define PIPE_YPOS_CIRCLE_EDGE_ID 5

#define PIPE_RADIUS 0.3f
#define EXT_NORMAL vec3(1.0, 0, 0)
#define EXT_CENTER vec3(-1, 0.5, 0.0)


__device__ vec3 snapToPipeSurface(vec3 pos)
{
    vec3 center;

    if(pos.x < 0.5) // Straights
    {
        center = vec3(pos.x, (pos.y < 0.0 ? -0.5 : 0.5), 0.0);
    }
    else // Arc
    {
        center = pos - vec3(0.5, 0.0, pos.z);
        center = normalize(center) * 0.5f;
        center = vec3(0.5, 0, 0) + center;
    }

    vec3 dist = pos - center;
    vec3 extProj = normalize(dist) * PIPE_RADIUS;
    return center + extProj;
}

__device__ vec3 snapToPipeExtremityFace(vec3 pos)
{
    vec3 center = EXT_CENTER;
    center.y *= sign(pos.y);

    float offset = dot(pos - center, EXT_NORMAL);
    return pos - EXT_NORMAL * offset;
}

__device__ vec3 snapToPipeExtremityEdge(vec3 pos)
{
    vec3 center = EXT_CENTER;
    center.y *= sign(pos.y);

    vec3 dist = pos - center;
    float offset = dot(dist, EXT_NORMAL);
    vec3 extProj = dist - EXT_NORMAL * offset;
    return center + normalize(extProj) * PIPE_RADIUS;
}

__device__ vec3 pipeSnapToBoundary(int boundaryID, vec3 pos)
{
    switch(boundaryID)
    {
    case PIPE_CYLINDER_FACE_ID :
        return snapToPipeSurface(pos);
    case PIPE_YNEG_DISK_FACE_ID :
    case PIPE_YPOS_DISK_FACE_ID :
        return snapToPipeExtremityFace(pos);
    case PIPE_YNEG_CIRCLE_EDGE_ID :
    case PIPE_YPOS_CIRCLE_EDGE_ID :
        return snapToPipeExtremityEdge(pos);
    }

    return pos;
}


__device__ snapToBoundaryFct pipeSnapToBoundaryPtr = pipeSnapToBoundary;


// CUDA Drivers
void installCudaPipeBoundary()
{
    snapToBoundaryFct d_snapToBoundary = nullptr;
    hipMemcpyFromSymbol(&d_snapToBoundary, HIP_SYMBOL(pipeSnapToBoundaryPtr), sizeof(snapToBoundaryFct));
    hipMemcpyToSymbol(HIP_SYMBOL(snapToBoundary), &d_snapToBoundary, sizeof(snapToBoundaryFct));


    if(verboseCuda)
        printf("I -> CUDA \tPipe boundary installed\n");
}
