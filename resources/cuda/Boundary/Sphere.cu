#include "Base.cuh"

__device__ vec3 sphereSnapToBoundary(int boundaryID, vec3 pos)
{
    vec3 npos = normalize(pos);
    if(boundaryID == 1)
        return npos * 0.5f;
    else
        return npos;
}


__device__ snapToBoundaryFct sphereSnapToBoundaryPtr = sphereSnapToBoundary;


// CUDA Drivers
void installCudaSphereBoundary()
{
    snapToBoundaryFct d_snapToBoundary = nullptr;
    hipMemcpyFromSymbol(&d_snapToBoundary, HIP_SYMBOL(sphereSnapToBoundaryPtr), sizeof(snapToBoundaryFct));
    hipMemcpyToSymbol(HIP_SYMBOL(snapToBoundary), &d_snapToBoundary, sizeof(snapToBoundaryFct));


    if(verboseCuda)
        printf("I -> CUDA \tSphere boundary installed\n");
}
