#include "Base.cuh"

__device__ vec3 sphereSnapToBoundary(int boundaryID, vec3 pos)
{
    return normalize(pos);
}


__device__ snapToBoundaryFct sphereSnapToBoundaryPtr = sphereSnapToBoundary;


// CUDA Drivers
void installCudaSphereBoundary()
{
    snapToBoundaryFct d_snapToBoundary = nullptr;
    hipMemcpyFromSymbol(&d_snapToBoundary, HIP_SYMBOL(sphereSnapToBoundaryPtr), sizeof(snapToBoundaryFct));
    hipMemcpyToSymbol(HIP_SYMBOL(snapToBoundary), &d_snapToBoundary, sizeof(snapToBoundaryFct));


    if(verboseCuda)
        printf("I -> CUDA \tSphere boundary installed\n");
}
