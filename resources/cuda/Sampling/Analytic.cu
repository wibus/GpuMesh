#include "Base.cuh"


__device__ mat3 vertMetric(const vec3& position);

__device__ mat3 analyticMetricAt(const vec3& position)
{
    return vertMetric(position);
}

__device__ metricAtFct analyticMetricAtPtr = analyticMetricAt;


// CUDA Drivers
void installCudaAnalyticSampler()
{
    metricAtFct d_metricAt = nullptr;
    hipMemcpyFromSymbol(&d_metricAt, HIP_SYMBOL(analyticMetricAtPtr), sizeof(metricAtFct));
    hipMemcpyToSymbol(HIP_SYMBOL(metricAt), &d_metricAt, sizeof(metricAtFct));

    printf("I -> CUDA \tAnalytic Discritizer installed\n");
}
