#include "Base.cuh"


__device__ mat3 vertMetric(const vec3& position);

__device__ mat3 analyticMetricAt(const vec3& position, uint cacheId)
{
    return vertMetric(position);
}

__device__ metricAtFct analyticMetricAtPtr = analyticMetricAt;


// CUDA Drivers
void installCudaAnalyticSampler()
{
    metricAtFct d_metricAt = nullptr;
    hipMemcpyFromSymbol(&d_metricAt, HIP_SYMBOL(analyticMetricAtPtr), sizeof(metricAtFct));
    hipMemcpyToSymbol(HIP_SYMBOL(metricAt), &d_metricAt, sizeof(metricAtFct));


    if(verboseCuda)
        printf("I -> CUDA \tAnalytic Discritizer installed\n");
}
