#include "Base.cuh"


__device__ mat3 localMetricAt(const vec3& position)
{
    return mat3(0.0);
}

__device__ metricAtFct localMetricAtPtr = localMetricAt;


// CUDA Drivers
void installCudaLocalSampler()
{
    metricAtFct d_metricAt = nullptr;
    hipMemcpyFromSymbol(&d_metricAt, HIP_SYMBOL(localMetricAtPtr), sizeof(metricAtFct));
    hipMemcpyToSymbol(HIP_SYMBOL(metricAt), &d_metricAt, sizeof(metricAtFct));

    printf("I -> CUDA \tLocal Discritizer installed\n");
}
