#include "DataStructures/GpuMesh.h"

#include "Mesh.cuh"



///////////////////////
// Mesh data buffers //
///////////////////////
__device__ Vert* verts = nullptr;
__device__ uint verts_length = 0;

__device__ Tet* tets = nullptr;
__device__ uint tets_length = 0;

__device__ Pri* pris = nullptr;
__device__ uint pris_length = 0;

__device__ Hex* hexs = nullptr;
__device__ uint hexs_length = 0;

__device__ Topo* topos = nullptr;
__device__ uint topos_length = 0;

__device__ NeigVert* neigVerts = nullptr;
__device__ uint neigVerts_length = 0;

__device__ NeigElem* neigElems = nullptr;
__device__ uint neigElems_length = 0;

__device__ uint* groupMembers = nullptr;
__device__ uint groupMembers_length = 0;


// CUDA Drivers
size_t d_vertsLength = 0;
GpuVert* d_verts = nullptr;
void updateCudaVerts(const GpuVert* vertsBuff, size_t vertsLength)
{
    size_t vertsBuffSize = sizeof(GpuVert) * vertsLength;
    if(d_verts == nullptr || d_vertsLength != vertsLength)
    {
        hipFree(d_verts);
        hipMalloc(&d_verts, vertsBuffSize);
        hipMemcpyToSymbol(HIP_SYMBOL(verts), &d_verts, sizeof(d_verts));

        d_vertsLength = vertsLength;
        hipMemcpyToSymbol(HIP_SYMBOL(verts_length), &vertsLength, sizeof(uint));
    }

    hipMemcpy(d_verts, vertsBuff, vertsBuffSize, hipMemcpyHostToDevice);
    printf("I -> CUDA \tverts updated\n");
}


size_t d_tetLength = 0;
GpuTet* d_tets = nullptr;
void updateCudaTets(const std::vector<GpuTet>& tetBuff)
{
    uint tetLength = tetBuff.size();
    size_t tetBuffSize = sizeof(decltype(tetBuff.front())) * tetLength;
    if(d_tets == nullptr || d_tetLength != tetLength)
    {
        hipFree(d_tets);
        hipMalloc(&d_tets, tetBuffSize);
        hipMemcpyToSymbol(HIP_SYMBOL(tets), &d_tets, sizeof(d_tets));

        d_tetLength = tetLength;
        hipMemcpyToSymbol(HIP_SYMBOL(tets_length), &tetLength, sizeof(uint));
    }

    hipMemcpy(d_tets, tetBuff.data(), tetBuffSize, hipMemcpyHostToDevice);
    printf("I -> CUDA \ttets updated\n");
}


size_t d_priLength = 0;
GpuPri* d_pris = nullptr;
void updateCudaPris(const std::vector<GpuPri>& priBuff)
{
    uint priLength = priBuff.size();
    size_t priBuffSize = sizeof(decltype(priBuff.front())) * priLength;
    if(d_pris == nullptr || d_priLength != priLength)
    {
        hipFree(d_pris);
        hipMalloc(&d_pris, priBuffSize);
        hipMemcpyToSymbol(HIP_SYMBOL(pris), &d_pris, sizeof(d_pris));

        d_priLength = priLength;
        hipMemcpyToSymbol(HIP_SYMBOL(pris_length), &priLength, sizeof(uint));
    }

    hipMemcpy(d_pris, priBuff.data(), priBuffSize, hipMemcpyHostToDevice);
    printf("I -> CUDA \tpris updated\n");
}


size_t d_hexLength = 0;
GpuHex* d_hexs = nullptr;
void updateCudaHexs(const std::vector<GpuHex>& hexBuff)
{
    uint hexLength = hexBuff.size();
    size_t hexBuffSize = sizeof(decltype(hexBuff.front())) * hexLength;
    if(d_hexs == nullptr || d_hexLength != hexLength)
    {
        hipFree(d_hexs);
        hipMalloc(&d_hexs, hexBuffSize);
        hipMemcpyToSymbol(HIP_SYMBOL(hexs), &d_hexs, sizeof(d_hexs));

        d_hexLength = hexLength;
        hipMemcpyToSymbol(HIP_SYMBOL(hexs_length), &hexLength, sizeof(uint));
    }

    hipMemcpy(d_hexs, hexBuff.data(), hexBuffSize, hipMemcpyHostToDevice);
    printf("I -> CUDA \thexs updated\n");
}


size_t d_topoLength = 0;
Topo* d_topos = nullptr;

size_t d_neighVertLength = 0;
NeigVert* d_neigVerts = nullptr;

size_t d_neighElemLength = 0;
NeigElem* d_neigElems = nullptr;

void updateCudaTopo(
        const std::vector<GpuTopo>& topoBuff,
        const std::vector<GpuNeigVert>& neigVertBuff,
        const std::vector<GpuNeigElem>& neigElemBuff)
{
    // Topologies
    uint topoLength = topoBuff.size();
    size_t topoBuffSize = sizeof(decltype(topoBuff.front())) * topoLength;
    if(d_topos == nullptr || d_topoLength != topoLength)
    {
        hipFree(d_topos);
        hipMalloc(&d_topos, topoBuffSize);
        hipMemcpyToSymbol(HIP_SYMBOL(topos), &d_topos, sizeof(d_topos));

        d_topoLength = topoLength;
        hipMemcpyToSymbol(HIP_SYMBOL(topos_length), &topoLength, sizeof(uint));
    }

    hipMemcpy(d_topos, topoBuff.data(), topoBuffSize, hipMemcpyHostToDevice);
    printf("I -> CUDA \ttopos updated\n");


    // Neighbor vertices
    uint neigVertLength = neigVertBuff.size();
    size_t neigVertBuffSize = sizeof(decltype(neigVertBuff.front())) * neigVertLength;
    if(d_neigVerts == nullptr || d_neighVertLength != neigVertLength)
    {
        hipFree(d_neigVerts);
        hipMalloc(&d_neigVerts, neigVertBuffSize);
        hipMemcpyToSymbol(HIP_SYMBOL(neigVerts), &d_neigVerts, sizeof(d_neigVerts));

        d_neighVertLength = neigVertLength;
        hipMemcpyToSymbol(HIP_SYMBOL(neigVerts_length), &neigVertLength, sizeof(uint));
    }

    hipMemcpy(d_neigVerts, neigVertBuff.data(), neigVertBuffSize, hipMemcpyHostToDevice);
    printf("I -> CUDA \tneigVerts updated\n");


    // Neighbor elements
    uint neigElemLength = neigElemBuff.size();
    size_t neigElemBuffSize = sizeof(decltype(neigElemBuff.front())) * neigElemLength;
    if(d_neigElems == nullptr || d_neighElemLength != neigElemLength)
    {
        hipFree(d_neigElems);
        hipMalloc(&d_neigElems, neigElemBuffSize);
        hipMemcpyToSymbol(HIP_SYMBOL(neigElems), &d_neigElems, sizeof(d_neigElems));

        d_neighElemLength = neigElemLength;
        hipMemcpyToSymbol(HIP_SYMBOL(neigElems_length), &neigElemLength, sizeof(uint));
    }

    hipMemcpy(d_neigElems, neigElemBuff.data(), neigElemBuffSize, hipMemcpyHostToDevice);
    printf("I -> CUDA \tneigElems updated\n");
}


size_t d_groupMembersLength = 0;
GLuint* d_groupMembers = nullptr;
void updateCudaGroupMembers(
        const std::vector<GLuint>& groupMemberBuff)
{
    // Group members
    uint groupMembersLength = groupMemberBuff.size();
    size_t groupMembersBuffSize = sizeof(decltype(groupMemberBuff.front())) * groupMembersLength;
    if(d_groupMembers == nullptr || d_groupMembersLength != groupMembersLength)
    {
        hipFree(d_groupMembers);
        hipMalloc(&d_groupMembers, groupMembersBuffSize);
        hipMemcpyToSymbol(HIP_SYMBOL(groupMembers), &d_groupMembers, sizeof(d_groupMembers));

        d_groupMembersLength = groupMembersLength;
        hipMemcpyToSymbol(HIP_SYMBOL(groupMembers_length), &groupMembersLength, sizeof(uint));
    }

    hipMemcpy(d_groupMembers, groupMemberBuff.data(), groupMembersBuffSize, hipMemcpyHostToDevice);
    printf("I -> CUDA \tgroupMembers updated\n");
}
