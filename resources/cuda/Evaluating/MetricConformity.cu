#include "hip/hip_runtime.h"
#include "Base.cuh"


#define TWO_I mat3(2.0)

#define Fr_TET_INV mat3( \
    vec3(1, 0, 0), \
    vec3(-0.5773502691896257645091, 1.154700538379251529018, 0), \
    vec3(-0.4082482904638630163662, -0.4082482904638630163662, 1.224744871391589049099))

#define Fr_PRI_INV mat3( \
    vec3(1.0, 0.0, 0.0), \
    vec3(-0.5773502691896257645091, 1.154700538379251529018, 0.0), \
    vec3(0.0, 0.0, 1.0))

__device__ mat3 specifiedMetric(
        const vec3& v0,
        const vec3& v1,
        const vec3& v2,
        const vec3& v3,
        uint& cachedRefTet)
{
    const float H = 0.5;
    const float Q = (1.0 - H) / 3.0;
    return ((*metricAt)((v0 + v1 + v2 + v3)/4.0f,  cachedRefTet) * (-0.8f) +
            (*metricAt)(v0*H + v1*Q + v2*Q + v3*Q, cachedRefTet) * 0.45f +
            (*metricAt)(v0*Q + v1*H + v2*Q + v3*Q, cachedRefTet) * 0.45f +
            (*metricAt)(v0*Q + v1*Q + v2*H + v3*Q, cachedRefTet) * 0.45f +
            (*metricAt)(v0*Q + v1*Q + v2*Q + v3*H, cachedRefTet) * 0.45f);
}

__device__ float metricConformity(const mat3& Fk, const mat3& Ms)
{
    mat3 Fk_inv = inverse(transpose(Fk));
    mat3 Mk = Fk_inv * transpose(Fk_inv);
    mat3 Mk_inv = inverse(Mk);
    mat3 Ms_inv = inverse(Ms);

    mat3 tNc = Mk_inv*Ms + Ms_inv*Mk - TWO_I;

    float tNc_frobenius2 =
            dot(tNc[0], tNc[0]) +
            dot(tNc[1], tNc[1]) +
            dot(tNc[2], tNc[2]);

    float Fk_sign = sign(determinant(Fk));
    return Fk_sign / (1.0 + sqrt(tNc_frobenius2));
}

__device__ float metricConformityTetQuality(const vec3 vp[TET_VERTEX_COUNT], const Tet& tet)
{
    vec3 e03 = vp[3] - vp[0];
    vec3 e13 = vp[3] - vp[1];
    vec3 e23 = vp[3] - vp[2];

    mat3 Fk = mat3(e03, e13, e23) * Fr_TET_INV;

    mat3 Ms0 = specifiedMetric(vp[0], vp[1], vp[2], vp[3], tet.c[0]);

    float qual0 = metricConformity(Fk, Ms0);

    return qual0;
}

__device__ float metricConformityPriQuality(const vec3 vp[PRI_VERTEX_COUNT], const Pri& pri)
{
    vec3 e03 = vp[3] - vp[0];
    vec3 e14 = vp[4] - vp[1];
    vec3 e25 = vp[5] - vp[2];
    vec3 e01 = vp[1] - vp[0];
    vec3 e12 = vp[2] - vp[1];
    vec3 e20 = vp[0] - vp[2];
    vec3 e34 = vp[4] - vp[3];
    vec3 e45 = vp[5] - vp[4];
    vec3 e53 = vp[3] - vp[5];

    // Prism corner quality is not invariant under edge swap
    // Third edge is the expected to be colinear with the first two's cross product
    mat3 Fk0 = mat3(-e01, e20, e03) * Fr_PRI_INV;
    mat3 Fk1 = mat3(-e12, e01, e14) * Fr_PRI_INV;
    mat3 Fk2 = mat3(-e20, e12, e25) * Fr_PRI_INV;
    mat3 Fk3 = mat3(-e34, e53, e03) * Fr_PRI_INV;
    mat3 Fk4 = mat3(-e45, e34, e14) * Fr_PRI_INV;
    mat3 Fk5 = mat3(-e53, e45, e25) * Fr_PRI_INV;

    mat3 Ms0 = specifiedMetric(vp[0], vp[1], vp[2], vp[3], pri.c[0]);
    mat3 Ms1 = specifiedMetric(vp[0], vp[1], vp[2], vp[4], pri.c[1]);
    mat3 Ms2 = specifiedMetric(vp[0], vp[1], vp[2], vp[5], pri.c[2]);
    mat3 Ms3 = specifiedMetric(vp[0], vp[3], vp[4], vp[5], pri.c[3]);
    mat3 Ms4 = specifiedMetric(vp[1], vp[3], vp[4], vp[5], pri.c[4]);
    mat3 Ms5 = specifiedMetric(vp[2], vp[3], vp[4], vp[5], pri.c[5]);

    float qual0 = metricConformity(Fk0, Ms0);
    float qual1 = metricConformity(Fk1, Ms1);
    float qual2 = metricConformity(Fk2, Ms2);
    float qual3 = metricConformity(Fk3, Ms3);
    float qual4 = metricConformity(Fk4, Ms4);
    float qual5 = metricConformity(Fk5, Ms5);

    return (qual0 + qual1 + qual2 + qual3 + qual4 + qual5) / 6.0;
}

__device__ float metricConformityHexQuality(const vec3 vp[HEX_VERTEX_COUNT], const Hex& hex)
{
    vec3 e01 = vp[1] - vp[0];
    vec3 e03 = vp[3] - vp[0];
    vec3 e04 = vp[4] - vp[0];
    vec3 e12 = vp[2] - vp[1];
    vec3 e15 = vp[5] - vp[1];
    vec3 e23 = vp[3] - vp[2];
    vec3 e26 = vp[6] - vp[2];
    vec3 e37 = vp[7] - vp[3];
    vec3 e45 = vp[5] - vp[4];
    vec3 e47 = vp[7] - vp[4];
    vec3 e56 = vp[6] - vp[5];
    vec3 e67 = vp[7] - vp[6];

    // Since hex's corner matrix is the identity matrix,
    // there's no need to define Fr_INV.
    mat3 Fk0 = mat3(e01,  e04, -e03);
    mat3 Fk1 = mat3(e01,  e12,  e15);
    mat3 Fk2 = mat3(e12,  e26, -e23);
    mat3 Fk3 = mat3(e03,  e23,  e37);
    mat3 Fk4 = mat3(e04,  e45,  e47);
    mat3 Fk5 = mat3(e15, -e56,  e45);
    mat3 Fk6 = mat3(e26,  e56,  e67);
    mat3 Fk7 = mat3(e37,  e67, -e47);

    mat3 Ms0 = specifiedMetric(vp[0], vp[1], vp[3], vp[4], hex.c[0]);
    mat3 Ms1 = specifiedMetric(vp[0], vp[1], vp[2], vp[5], hex.c[1]);
    mat3 Ms2 = specifiedMetric(vp[1], vp[2], vp[3], vp[6], hex.c[2]);
    mat3 Ms3 = specifiedMetric(vp[0], vp[2], vp[3], vp[7], hex.c[3]);
    mat3 Ms4 = specifiedMetric(vp[0], vp[4], vp[5], vp[7], hex.c[4]);
    mat3 Ms5 = specifiedMetric(vp[1], vp[4], vp[5], vp[6], hex.c[5]);
    mat3 Ms6 = specifiedMetric(vp[2], vp[5], vp[6], vp[7], hex.c[6]);
    mat3 Ms7 = specifiedMetric(vp[3], vp[4], vp[6], vp[7], hex.c[7]);

    float qual0 = metricConformity(Fk0, Ms0);
    float qual1 = metricConformity(Fk1, Ms1);
    float qual2 = metricConformity(Fk2, Ms2);
    float qual3 = metricConformity(Fk3, Ms3);
    float qual4 = metricConformity(Fk4, Ms4);
    float qual5 = metricConformity(Fk5, Ms5);
    float qual6 = metricConformity(Fk6, Ms6);
    float qual7 = metricConformity(Fk7, Ms7);

    return (qual0 + qual1 + qual2 + qual3 + qual4 + qual5 + qual6 + qual7) / 8.0;
}

__device__ tetQualityFct metricConformityTetQualityPtr = metricConformityTetQuality;
__device__ priQualityFct metricConformityPriQualityPtr = metricConformityPriQuality;
__device__ hexQualityFct metricConformityHexQualityPtr = metricConformityHexQuality;


// CUDA Drivers
void installCudaMetricConformityEvaluator()
{
    tetQualityFct d_tetQuality = nullptr;
    hipMemcpyFromSymbol(&d_tetQuality, HIP_SYMBOL(metricConformityTetQualityPtr), sizeof(tetQualityFct));
    hipMemcpyToSymbol(HIP_SYMBOL(tetQualityImpl), &d_tetQuality, sizeof(tetQualityFct));

    priQualityFct d_priQuality = nullptr;
    hipMemcpyFromSymbol(&d_priQuality, HIP_SYMBOL(metricConformityPriQualityPtr), sizeof(priQualityFct));
    hipMemcpyToSymbol(HIP_SYMBOL(priQualityImpl), &d_priQuality, sizeof(priQualityFct));

    hexQualityFct d_hexQuality = nullptr;
    hipMemcpyFromSymbol(&d_hexQuality, HIP_SYMBOL(metricConformityHexQualityPtr), sizeof(hexQualityFct));
    hipMemcpyToSymbol(HIP_SYMBOL(hexQualityImpl), &d_hexQuality, sizeof(hexQualityFct));


    if(verboseCuda)
        printf("I -> CUDA \tMetric Conformity Evaluator installed\n");
}
